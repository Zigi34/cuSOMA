#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

//FUNCTIONS
#define imin(a,b) (a<b?a:b)
#define imax(a,b) (a<b?b:a)

//#define SHARED 32

//POPULATION
#define NUM_VALS_BLOCK 64
#define NUM_VALS 256
#define DIMENSION 1024

//THREADS, BLOCKS
#define THREADS 64
#define FITNESS_THREADS 128
#define BLOCKS imin(8192, (NUM_VALS_BLOCK+THREADS-1)/THREADS)
#define FITNESS_BLOCKS imin(8192, (NUM_VALS+FITNESS_THREADS-1)/FITNESS_THREADS)
#define PRT_BLOCKS imin(8192, (DIMENSION+DIMENSION-1)/THREADS)

//SOLUTION
#define RANDOM_MAXIMUM 500

//CONFIG
#define CONF_PARMS 2
#define CONF_PATHLENGTH 0
#define CONF_STEP 1

#define ITERATION 1000

//LEADER
//__constant__ best_indiv leader[1];
__constant__ float config[DIMENSION];

FILE * pFile;

void log(const char* type, float time) {
	fprintf(pFile, "(%s) elapsed: %1.3f\n", type, time);
}

//DEBUG ERRORS
#pragma region ERROR FUNCTION
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#pragma endregion

#pragma region REDUCTION

__device__ float atomicMaxf(float* address, float val)
{
	int *address_as_int = (int*)address;
	int old = *address_as_int, assumed;
	while (val > __int_as_float(old)) {
		assumed = old;
		old = atomicCAS(address_as_int, assumed,
			__float_as_int(val));
	}
	return __int_as_float(old);
}
__global__ void max_reduce(const float* values, int* max_index, float* maximum, 
	const size_t elements)
{
	__shared__ float shared[FITNESS_THREADS];

	int tid = threadIdx.x;
	int gid = (blockDim.x * blockIdx.x) + tid;
	shared[tid] = 0.0f;

	if (gid < elements)
		shared[tid] = values[gid];
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s && gid < elements)
			shared[tid] = max(shared[tid], shared[tid + s]);  // 2
		__syncthreads();
	}
	
	__syncthreads();
	if (tid == 0) {
		maximum[blockIdx.x] = shared[tid];
		atomicMaxf(maximum, shared[0]);
	}	
	__syncthreads();
	if (gid < elements && maximum[0] == values[gid]) {
		__syncthreads();
		if (gid < elements) {
			*max_index = gid;
		}
	}
}

#pragma endregion

#pragma region RANDOM GENERATOR
__global__ void init_rand(hiprandState *state, size_t size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		hiprand_init(1337, idx, 0, &state[idx]);
}
__global__ void make_rand(hiprandState *state, float
	*randArray, const size_t size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		randArray[idx] = hiprand_uniform(&state[idx]);
}
#pragma endregion

#pragma region UTIL FUNCTION
float random_float()
{
	return (float)(rand() % RANDOM_MAXIMUM) + 1.0;
}
float random_int()
{
	return (int)(rand() % RANDOM_MAXIMUM) + 1.0;
}
float max_value(float* values, const size_t size) {
	float vysledek = 0.0;
	for (int i = 0; i<size; i++) {
		if (values[i] > vysledek)
			vysledek = values[i];
	}
	return vysledek;
}
#pragma endregion

#pragma region POPULATION
void population_fill(float* arr, const size_t size)
{
	int index = 0;
	int all = size;

	int result = imin(NUM_VALS_BLOCK, all);
	all -= NUM_VALS_BLOCK;

	while (result > 0) {
		for (int i = 0; i < DIMENSION; ++i) {
			for (int j = 0; j < result; j++)
				arr[index++] = random_float();
		}
		result = imin(NUM_VALS_BLOCK, all);
		all -= result;
	}
}
#pragma endregions

#pragma region SOMA EVOLUTION
/*
__global__ void step(const individual* pop, individual* new_pop, float* prt, size_t size) {
	int lid = threadIdx.x;
	int gid = blockIdx.x * THREADS + threadIdx.x;
	
	//for (float j = config[CONF_STEP]; j < config[CONF_PATHLENGTH]; j += config[CONF_STEP]) {
		#pragma unroll
		for (int i = 0; i < DIMENSION; i++) {
			if (prt[i] > 0.5) {
				new_pop[gid].val[i] = (leader[0].val[i] - pop[gid].val[i]) * config[CONF_STEP] + pop[gid].val[i];
			} else {
				new_pop[gid].val[i] = pop[gid].val[i];
			}
		}
	//}
	__syncthreads();
}
__global__ void evaluate(individual* individuals, const size_t size)
{
	int tid = blockIdx.x * THREADS + threadIdx.x;
	while (tid < size)
	{
		individuals[tid].fitness = (individuals[tid].val[0] * individuals[tid].val[1] * individuals[tid].val[2]);
		tid += THREADS * BLOCKS;
	}
	__syncthreads();
}
*/
/*
void soma(float *values, const size_t size)
{
	dim3 blocks(BLOCKS, 1);
	dim3 threads(THREADS, 1);
	
	//DEVICE
	float* dev_fitness;
	best_indiv* dev_leader;
	float* dev_prt_vector;
	hiprandState* dev_states;
	individual* dev_indiv;
	individual* dev_new_pop;

	float host_config[CONF_PARMS];
	host_config[CONF_PATHLENGTH] = 10.0;
	host_config[CONF_STEP] = 1.1;

	//int size = s * (int)(host_config[CONF_PATHLENGTH] / host_config[CONF_STEP]);

	gpuErrchk(hipMalloc((void**)&dev_fitness, size * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_leader, sizeof(best_indiv)));
	gpuErrchk(hipMalloc((void**)&dev_new_pop, size * sizeof(individual)));

	best_indiv* host_leader = (best_indiv*)malloc(sizeof(best_indiv));
	individual* population = (individual*)malloc(size * sizeof(individual));

	#pragma region SOMA PARAMETER INITIALIZE
	hipMemcpyToSymbol(HIP_SYMBOL(config), host_config, sizeof(float)* CONF_PARMS);
	#pragma endregion

	#pragma region INITIALIZE
	gpuErrchk(hipMalloc((void**)&dev_prt_vector, DIMENSION * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_states, DIMENSION * sizeof(hiprandState)));

	init_rand << <1, threads >> >(dev_states, DIMENSION);
	make_rand << <1, threads >> >(dev_states, dev_prt_vector, DIMENSION);
	#pragma endregion
	
	#pragma region POPULATION TO GPU
	hipMalloc((void**)&dev_indiv, size * sizeof(individual));
	hipMemcpy(dev_indiv, values, size * sizeof(individual), hipMemcpyHostToDevice);
	#pragma endregion

	hipEvent_t startEvent, stopEvent;
	float elapsedTime;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	float sumEval = 0.0;
	float sumReduce = 0.0;
	float sumRandom = 0.0;
	float sumStep = 0.0;

	for (int iter = 0; iter < ITERATION; iter++) {

		hipEventRecord(startEvent, 0);
		evaluate << <blocks, threads >> >(dev_indiv, size);
		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);
		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		log("evaluat", elapsedTime);
		sumEval += elapsedTime;

		hipEventRecord(startEvent, 0);
		max_reduce << <blocks, threads >> >(dev_indiv, dev_fitness, dev_leader, size);
		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);
		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		log("reduce", elapsedTime);
		sumReduce += elapsedTime;
		gpuErrchk(hipMemcpy(host_leader, dev_leader, sizeof(best_indiv), hipMemcpyDeviceToHost));

		//printf("Leader: (%1.3f, %1.3f, %1.3f)(%d)\n", host_leader[0].val[0], host_leader[0].val[1], host_leader[0].val[2], host_leader[0].index);
		hipMemcpyToSymbol(HIP_SYMBOL(leader), host_leader, sizeof(best_indiv));

		hipEventRecord(startEvent, 0);
		make_rand << <1, threads >> >(dev_states, dev_prt_vector, DIMENSION);
		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);
		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		log("random", elapsedTime);
		sumRandom += elapsedTime;
		
		hipEventRecord(startEvent, 0);
		step << <blocks, threads >> >(dev_indiv, dev_new_pop, dev_prt_vector, size);
		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);
		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		log("step", elapsedTime);
		sumStep += elapsedTime;

		gpuErrchk(hipMemcpy(population, dev_indiv, size * sizeof(individual), hipMemcpyDeviceToHost));
		//for (int i = 0; i < 10; i++)
			//printf("%1.3f, %1.3f, %1.3f = %1.3f\n", population[i].val[0], population[i].val[1], population[i].val[2], population[i].fitness);
		//printf("\n");

		individual * temp = dev_new_pop;
		dev_new_pop = dev_indiv;
		dev_indiv = temp;
	}

	printf("Avg Evaluated: %1.3f\n", sumEval / ITERATION);
	printf("Avg Reduced: %1.3f\n", sumReduce / ITERATION);
	printf("Avg Random: %1.3f\n", sumRandom / ITERATION);
	printf("Avg Step: %1.3f\n", sumStep / ITERATION);
	
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	free(host_leader);
	free(population);

	hipFree(dev_fitness);
	hipFree(dev_leader);
	hipFree(dev_prt_vector);
	hipFree(dev_states);
	hipFree(dev_indiv);
	hipFree(dev_new_pop);
}
*/
#pragma endregion

__global__ void operace(float* in, float* out) {
	__shared__ float temp[NUM_VALS_BLOCK];

	unsigned int gid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int lid = threadIdx.x;
	temp[lid] = 0;	//copy to shared
	
	if (gid < NUM_VALS_BLOCK) {
		#pragma unroll
		for (unsigned int i = 0; i < DIMENSION * NUM_VALS_BLOCK; i += NUM_VALS_BLOCK)
		{
			temp[lid] += in[gid + i];
		}
		out[gid] = temp[lid];
	}
	__syncthreads();
}

bool test(float* values, unsigned int size) {
	float test = values[0];
	for (int i = 1; i < size; i++)
	if (values[i] != test)
		return false;
	return true;
}

int main(int argc, char *argv[])
{
	pFile = fopen("log.txt", "a");
	srand(time(NULL));
	
	int real_size = NUM_VALS;
	int max_size = imin(NUM_VALS_BLOCK, real_size);
	float elapsedTime = 0.0, elapsedMemcpy = 0.0, elapsedKernel = 0.0;
	float *host_in, *host_out;
	hipEvent_t startEvent, stopEvent;
	
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	printf("Velikost populace: %d\n", real_size);
	printf("Max size = %d\n", max_size);

	hipHostMalloc(&host_in, real_size * DIMENSION * sizeof(float));
	hipHostMalloc(&host_out, real_size * sizeof(float));
	
	population_fill(host_in, real_size);
	real_size -= max_size;

	float *device_in, *device_out, *device_maximum;
	float* host_leader = (float*)malloc(sizeof(float) * DIMENSION);
	int* device_leader_index;
	float* host_in_start = host_in;
	float* host_out_start = host_out;
	float* dev_prt_vector;
	hiprandState* dev_states;

	gpuErrchk(hipMalloc((void**)&device_in, max_size * DIMENSION * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&device_out, max_size * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&device_leader_index, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&device_maximum, FITNESS_BLOCKS * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_prt_vector, DIMENSION * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_states, DIMENSION * sizeof(hiprandState)));

	while (max_size > 0) {
		hipEventRecord(startEvent, 0);
		gpuErrchk(hipMemcpy(device_in, host_in_start, sizeof(float)* DIMENSION * max_size, hipMemcpyHostToDevice));
		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);
		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		elapsedMemcpy += elapsedTime;

		hipEventRecord(startEvent, 0);
		operace<< <BLOCKS, THREADS >> >(device_in, device_out);
		hipEventRecord(stopEvent, 0);
		hipEventSynchronize(stopEvent);
		hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
		elapsedKernel += elapsedTime;
	
		gpuErrchk(hipMemcpy(host_out_start, device_out, sizeof(float)* max_size, hipMemcpyDeviceToHost));

		host_in_start += max_size*DIMENSION;
		host_out_start += max_size;
		max_size = imin(real_size, NUM_VALS_BLOCK);
		real_size -= max_size;
	}
	
	printf("Memcpy: %1.3f\n", elapsedMemcpy);
	printf("Kernel: %1.3f\n", elapsedKernel);
	
	for (int i = 0; i < imin(NUM_VALS, 16); i++)
		printf("%1.3f\n", host_out[i]);
	
	hipEventRecord(startEvent, 0);
	max_reduce << <FITNESS_BLOCKS, FITNESS_THREADS >> >(device_out, device_leader_index, device_maximum, NUM_VALS);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Redukce: %1.3f\n", elapsedTime);
	
	int* host_leader_index = (int*)malloc(sizeof(int));
	gpuErrchk(hipMemcpy(host_leader_index, device_leader_index, sizeof(int), hipMemcpyDeviceToHost));

	int index = 0;
	for (int i = host_leader_index[0]; i < DIMENSION * NUM_VALS; i += DIMENSION) {
		host_leader[index++] = host_in[i];
	}

	hipMemcpyToSymbol(HIP_SYMBOL(config), host_leader, sizeof(float)* DIMENSION);

	hipEventRecord(startEvent, 0);
	init_rand << <PRT_BLOCKS, THREADS >> >(dev_states, DIMENSION);
	make_rand << <PRT_BLOCKS, THREADS >> >(dev_states, dev_prt_vector, DIMENSION);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	printf("Random: %1.3f\n", elapsedTime);

	printf("Maximum na indexu %d\n", host_leader_index[0]);
	
	hipFree(device_in);
	hipFree(device_out);
	hipFree(device_leader_index);
	hipFree(device_maximum);
	hipFree(dev_prt_vector);
	hipFree(dev_states);
	
	hipHostFree(host_in);
	hipHostFree(host_out);
	
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);

	free(host_leader);
	free(host_leader_index);
	fclose(pFile);
}

