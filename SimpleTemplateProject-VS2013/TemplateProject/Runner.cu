#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

//FUNCTIONS
#define imin(a,b) (a<b?a:b)
#define imax(a,b) (a<b?b:a)

//THREADS, BLOCKS
#define THREADS 64
#define BLOCKS imin(1024, (NUM_VALS+THREADS-1)/THREADS)

//POPULATION
#define NUM_VALS 8192
#define DIMENSION 3

//SOLUTION
#define RANDOM_MAXIMUM 500

//CONFIG
#define CONF_PARMS 2
#define CONF_PATHLENGTH 0
#define CONF_STEP 1

#pragma region SOLUTION
typedef struct __align__(16) individual{
	float val[DIMENSION];
	float fitness;
} individual;
#pragma endregion

typedef struct __align__(16) best_indiv {
	float val[DIMENSION];
	unsigned int index;
} best_indiv;


//LEADER
__constant__ best_indiv leader[1];
__constant__ float config[CONF_PARMS];

//DEBUG ERRORS
#pragma region ERROR FUNCTION
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#pragma endregion

#pragma region REDUCTION
__device__ float atomicMaxf(float* address, float val)
{
	int *address_as_int = (int*)address;
	int old = *address_as_int, assumed;
	while (val > __int_as_float(old)) {
		assumed = old;
		old = atomicCAS(address_as_int, assumed,
			__float_as_int(val));
	}
	return __int_as_float(old);
}
__global__ void max_reduce(const individual* const d_array, float* d_max, best_indiv* leader,
	const size_t elements)
{
	__shared__ float shared[THREADS];

	int tid = threadIdx.x;
	int gid = (blockDim.x * blockIdx.x) + tid;
	shared[tid] = 0.0f;

	if (gid < elements)
		shared[tid] = d_array[gid].fitness;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s && gid < elements)
			shared[tid] = max(shared[tid], shared[tid + s]);  // 2
		__syncthreads();
	}
	// what to do now?
	// option 1: save block result and launch another kernel
	if (tid == 0)
		d_max[blockIdx.x] = shared[tid]; // 3
	// option 2: use atomics
	if (tid == 0)
		atomicMaxf(d_max, shared[0]);
	__syncthreads();
	
	if (gid < elements && d_max[0] == d_array[gid].fitness) {
		for (int i = 0; i < DIMENSION; i++)
			leader[0].val[i] = d_array[gid].val[i];
		leader[0].index = gid;
	}
}
#pragma endregion

#pragma region RANDOM GENERATOR
__global__ void init_rand(hiprandState *state, size_t size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		hiprand_init(1337, idx, 0, &state[idx]);
}
__global__ void make_rand(hiprandState *state, float
	*randArray, const size_t size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		randArray[idx] = hiprand_uniform(&state[idx]);
}
#pragma endregion

#pragma region RANDOM POPULATION
__global__ void random_population(hiprandState *state, individual* pop, const size_t size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		for (int i = 0; i < DIMENSION; i++)
			pop[idx].val[i] = hiprand_uniform(&state[idx]) * RANDOM_MAXIMUM;
	}
}
#pragma endregion

#pragma region UTIL FUNCTION
void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);
}
float random_float()
{
	return (float)(rand() % RANDOM_MAXIMUM) + 1.0;
}
void individual_print(individual *population, const size_t size)
{
	int i;
	for (i = 0; i < NUM_VALS; ++i) {
		printf("(%1.1f, %1.1f, %1.1f) -> %1.1f\n ", population[i].val[0], population[i].val[1], population[i].val[2], population[i].fitness);
	}
	printf("\n");
}
void population_fill(individual *arr, const size_t size)
{
	srand(time(NULL));
	for (int i = 0; i < size; ++i) {
		arr[i].val[0] = random_float();
		arr[i].val[1] = random_float();
		arr[i].val[2] = random_float();
		arr[i].fitness = 0.0;
	}
}
float max_value(individual* values, const size_t size) {
	float vysledek = 0.0;
	for (int i = 0; i<size; i++) {
		if (values[i].fitness > vysledek)
			vysledek = values[i].fitness;
	}
	return vysledek;
}
#pragma endregion

#pragma region SOMA EVOLUTION
__global__ void step(const individual* pop, individual* new_pop, float* prt, size_t size) {
	int lid = threadIdx.x;
	int gid = blockIdx.x * THREADS + threadIdx.x;
	
	//for (float j = config[CONF_STEP]; j < config[CONF_PATHLENGTH]; j += config[CONF_STEP]) {
		#pragma unroll
		for (int i = 0; i < DIMENSION; i++) {
			if (prt[i] > 0.5) {
				new_pop[gid].val[i] = (leader[0].val[i] - pop[gid].val[i]) * config[CONF_STEP] + pop[gid].val[i];
			} else {
				new_pop[gid].val[i] = pop[gid].val[i];
			}
		}
	//}
	__syncthreads();
}
__global__ void evaluate(individual* individuals, const size_t size)
{
	int tid = blockIdx.x * THREADS + threadIdx.x;
	while (tid < size)
	{
		individuals[tid].fitness = (individuals[tid].val[0] * individuals[tid].val[1] * individuals[tid].val[2]);
		tid += THREADS * BLOCKS;
	}
	__syncthreads();
}
void soma(individual *values, const size_t size)
{
	dim3 blocks(BLOCKS, 1);
	dim3 threads(THREADS, 1);
	
	//DEVICE
	float* dev_fitness;
	best_indiv* dev_leader;
	float* dev_prt_vector;
	hiprandState* dev_states;
	individual* dev_indiv;
	individual* dev_new_pop;

	float host_config[CONF_PARMS];
	host_config[CONF_PATHLENGTH] = 10.0;
	host_config[CONF_STEP] = 1.1;

	//int size = s * (int)(host_config[CONF_PATHLENGTH] / host_config[CONF_STEP]);

	gpuErrchk(hipMalloc((void**)&dev_fitness, size * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_leader, sizeof(best_indiv)));
	gpuErrchk(hipMalloc((void**)&dev_new_pop, size * sizeof(individual)));

	best_indiv* host_leader = (best_indiv*)malloc(sizeof(best_indiv));
	individual* population = (individual*)malloc(size * sizeof(individual));

	#pragma region SOMA PARAMETER INITIALIZE

	hipMemcpyToSymbol(HIP_SYMBOL(config), host_config, sizeof(float)* CONF_PARMS);
	#pragma endregion

	#pragma region INITIALIZE
	gpuErrchk(hipMalloc((void**)&dev_prt_vector, DIMENSION * sizeof(float)));
	gpuErrchk(hipMalloc((void**)&dev_states, DIMENSION * sizeof(hiprandState)));

	init_rand << <1, threads >> >(dev_states, DIMENSION);
	make_rand << <1, threads >> >(dev_states, dev_prt_vector, DIMENSION);
	#pragma endregion
	
	#pragma region POPULATION TO GPU
	hipMalloc((void**)&dev_indiv, size * sizeof(individual));
	hipMemcpy(dev_indiv, values, size * sizeof(individual), hipMemcpyHostToDevice);
	#pragma endregion

	for (int iter = 0; iter < 5000; iter++) {
		evaluate << <blocks, threads >> >(dev_indiv, size);
		max_reduce << <blocks, threads >> >(dev_indiv, dev_fitness, dev_leader, size);
		gpuErrchk(hipMemcpy(host_leader, dev_leader, sizeof(best_indiv), hipMemcpyDeviceToHost));

		//printf("Leader: (%1.3f, %1.3f, %1.3f)(%d)\n", host_leader[0].val[0], host_leader[0].val[1], host_leader[0].val[2], host_leader[0].index);
		hipMemcpyToSymbol(HIP_SYMBOL(leader), host_leader, sizeof(best_indiv));

		make_rand << <1, threads >> >(dev_states, dev_prt_vector, DIMENSION);
		step << <blocks, threads >> >(dev_indiv, dev_new_pop, dev_prt_vector, size);

		gpuErrchk(hipMemcpy(population, dev_indiv, size * sizeof(individual), hipMemcpyDeviceToHost));
		//for (int i = 0; i < 10; i++)
			//printf("%1.3f, %1.3f, %1.3f = %1.3f\n", population[i].val[0], population[i].val[1], population[i].val[2], population[i].fitness);
		//printf("\n");

		individual * temp = dev_new_pop;
		dev_new_pop = dev_indiv;
		dev_indiv = temp;
	}
	
	free(host_leader);
	free(population);

	hipFree(dev_fitness);
	hipFree(dev_leader);
	hipFree(dev_prt_vector);
	hipFree(dev_states);
	hipFree(dev_indiv);
	hipFree(dev_new_pop);
}

#pragma endregion


int main(int argc, char *argv[])
{
	//initializeCUDA(deviceProp);
	clock_t start, stop;
	size_t size = NUM_VALS;
	printf("Velikost populace: %d\n",size);

	individual *values = (individual*)malloc(size * sizeof(individual));
	population_fill(values, size);

	start = clock();
	soma(values, size); /* Inplace */
	stop = clock();

	//individual_print(values);
	//printf("MAX= %1.1f\n", max_value(values, NUM_VALS));
	free(values);
	print_elapsed(start, stop);
}

